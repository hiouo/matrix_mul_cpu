#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
#include <fstream>

// CUDA kernel for matrix multiplication
__global__ void matrixMulKernel(float* A, float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float value = 0;
        for (int k = 0; k < N; ++k) {
            value += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = value;
    }
}

void matrixMultiplyGPU(float* h_A, float* h_B, float* h_C, int N) {
    // The matrix size
    size_t size = N * N * sizeof(float);

    // Allocate device(GPU) memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy matrices from host(CPU) to device(GPU)
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Define block and grid dimensions
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Launch the matrix multiplication kernel
    matrixMulKernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();

    // Copy result from device to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main() {
    int N = 2000; // Example size of the matrix
    size_t size = N * N * sizeof(float);

    // Allocate host memory
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    // Initialize matrices with some values
    for (int i = 0; i < N * N; ++i) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(i);
    }

    // Perform matrix multiplication
    auto start = std::chrono::high_resolution_clock::now();
    matrixMultiplyGPU(h_A, h_B, h_C, N);
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float, std::milli> duration = end - start;

    // Write the first element of the result matrix to a file
    std::ofstream outputFile("gpu_output.txt");
    if (outputFile.is_open()) {
        outputFile << h_C[0] << " ";
        outputFile << "\n";
        outputFile.close();
    } else {
        std::cerr << "Unable to open file for writing\n";
    }

    std::cout << "finish" << std::endl;
    std::cout << "GPU Matrix multiplication took " << duration.count() << " ms\n";

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}